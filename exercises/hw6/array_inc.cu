#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

template <typename T>
void alloc_bytes(T &ptr, size_t num_bytes){

  hipMallocManaged(&ptr, num_bytes);
}

__global__ void inc(int *array, size_t n){
  size_t idx = threadIdx.x+blockDim.x*blockIdx.x;
  while (idx < n){
    array[idx]++;
    idx += blockDim.x*gridDim.x; // grid-stride loop
    }
}

const size_t  ds = 32ULL*1024ULL*1024ULL;

int main(){

  //int *h_array, *d_array;
  int *array;
  alloc_bytes(array, ds*sizeof(array[0]));
  //hipMalloc(&d_array, ds*sizeof(d_array[0]));
  cudaCheckErrors("hipMalloc Error");
  memset(array, 0, ds*sizeof(array[0]));
  //hipMemcpy(d_array, h_array, ds*sizeof(h_array[0]), hipMemcpyHostToDevice);
  //cudaCheckErrors("hipMemcpy H->D Error");
  int gpuDeviceId;
  hipGetDevice(&gpuDeviceId);
  hipMemPrefetchAsync(array, ds*sizeof(array[0]), gpuDeviceId);
  for(int i = 0; i < 10000; ++i) {
    inc<<<256, 256>>>(array, ds);
    hipDeviceSynchronize();
  }
  hipMemPrefetchAsync(array, ds*sizeof(array[0]), hipCpuDeviceId);
  cudaCheckErrors("kernel launch error");
  //hipMemcpy(h_array, d_array, ds*sizeof(h_array[0]), hipMemcpyDeviceToHost);
  cudaCheckErrors("kernel execution or hipMemcpy D->H Error");
  for (int i = 0; i < ds; i++) 
    if (array[i] != 10000) {printf("mismatch at %d, was: %d, expected: %d\n", i, array[i], 1); return -1;}
  printf("success!\n"); 
  return 0;
}
