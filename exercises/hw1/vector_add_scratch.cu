
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(double* a, double* b, double* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n)
        c[idx] = a[idx] + b[idx];
}

#define N 2

int main() {
    int size = sizeof(double) * N;

    double* a = (double*)malloc(size);
    double* b = (double*)malloc(size);
    double* c = (double*)malloc(size);

    a[0] = 0.840188;
    a[1] = 1.34;
    b[0] = 0.394383;
    b[1] = 0.43;

    double *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    vector_add<<<1,N>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; ++i) {
        printf("A[%d] = %f\n", i, a[i]);
        printf("B[%d] = %f\n", i, b[i]);
        printf("C[%d] = %f\n", i, c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(a);
    free(b);
    free(c);

    return 0;
}